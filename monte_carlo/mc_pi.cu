// import libs
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <hip/hip_runtime.h>
// curand for random points generation
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <hip/hip_runtime_api.h>

// Number of thread blocks in the grid
#define THREAD_BLOCKS 512
// Number of threads in a thread block
#define THREADS 2
// Number of random points
#define N 1000

// define a kernal to find area within the quarter circle
__global__ void mc_pi(float *d_input_x, float *d_input_y, int *d_area)
{
  // define index and find the value of x,y coordinates
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = d_input_x[index];
  int y = d_input_y[index];

  // increment d_area when a random point is located within the quarter circle
  if (x * x + y * y <= 1.0f)
  {
    atomicAdd(d_area, 1);
  }
}

int main(void)
{
  // step1: initialise GPU
  int deviceid = 0;
  int devCount = 0;
  hipGetDeviceCount(&devCount);
  if (deviceid < devCount)
  {
    hipSetDevice(deviceid);
  }
  else
    return 1;

  // step3: allocate memory in GPU
  size_t array_size = N * sizeof(float);
  float *d_input_x;
  float *d_input_y;
  int *d_area;

  hipMalloc((void **)&d_input_x, array_size);
  hipMalloc((void **)&d_input_y, array_size);
  hipMalloc((void **)&d_area, sizeof(int));

  // Step4: Random Number Generator
  hiprandGenerator_t gen;
  // Create random number generator
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  // Set the generator
  hiprandSetPseudoRandomGeneratorSeed(gen, 123456ULL);
  // generate the randoms
  hiprandGenerateUniform(gen, d_input_x, N);
  hiprandGenerateUniform(gen, d_input_y, N);

  // Step5: use the kernel to add area
  mc_pi<<<THREAD_BLOCKS, THREADS>>>(d_input_x, d_input_y, d_area);

  // Step6: copy the data from the host to the device & print results
  int *h_area = (int *)malloc(sizeof(int));
  hipMemcpy(h_area, d_area, sizeof(int), hipMemcpyDeviceToHost);

  printf("\nPi:\t%f\n", (4.0 * (*h_area)) / (float)N);

  return (0);
}