// random number generation
#include <hip/hip_runtime.h>
#include <hiprand.h>

// Declare variable
hiprandGenerator_t gen;

// Create random number generator
hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

// Set the generator options
curandSetPsuedoRandomGenerator(gen, 1234ULL);

// Generator the randoms
hiprandGenerateNormal(gen, d_input, NUM_ELS, 0.0f, 1.0f);
